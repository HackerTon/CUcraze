#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/device_functions.h>
#include "helper.cuh"

__global__ void add_3(float *stack, int number){
	int myid = blockIdx.x * blockDim.x + threadIdx.x;

	int new_id = myid + 1;

	temp[myid] = stack[myid];

	__syncthreads();

	extern __shared__ float temp[];

	if (myid < number){

		stack[myid] = temp[new_id] + temp[new_id - 1];

	}
}

int main(int argc, char** argv){

	//CUDA Event Function Declaration

	hipEvent_t start, stop;

	cuda(hipEventCreate(&start));
	cuda(hipEventCreate(&stop));

	float time = 0;

	int data_size = 512;

	float *dStack = 0;
	float *hStack = 0;

	cuda(hipMalloc(&dStack, sizeof(float)*data_size));
	cuda(hipHostMalloc(&hStack, sizeof(float)*data_size));

	for (int i = 0; i < data_size; i++){
		hStack[i] = i;
	}

	cuda(hipMemcpy(dStack, hStack, sizeof(float)*data_size, hipMemcpyHostToDevice));

	dim3 thread(data_size);
	dim3 block(1);

	cuda(hipEventRecord(start, 0));

	add_3<<<block, thread, sizeof(float)*data_size>>>(dStack, thread.x);

	cuda(hipEventRecord(stop, 0));

	cuda(hipEventSynchronize(stop));

	cuda(hipPeekAtLastError());

	cuda(hipEventElapsedTime(&time, start, stop));

	cuda(hipMemcpy(hStack, dStack, sizeof(float)*data_size, hipMemcpyDeviceToHost));

	for (int i = 0; i < data_size; i++){
		std::cout << hStack[i] << " space " << i << std::endl;
	}

	std::cout << "Time taken for the kernel to finish = " << time << "ms" << std::endl;

	cuda(hipFree(dStack));
	cuda(hipHostFree(hStack));
}
